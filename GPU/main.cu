#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <cstdint>
#include <vector>
#include <chrono>
#include <array>
#include <cmath>
#include "spline.hpp"
#include "my_defines.h"
#include "my_types.h"
#include "my_utils.h"

// To compile: nvcc -std=c++11 -Xcompiler -O2 -Xcompiler "-Wall -Wdouble-promotion -Wparentheses -Wconversion" main.cu -o main

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


#define CUDA_SAFE_CALL(__call) do {											\
  hipError_t __err = __call;												\
  if(__err != hipSuccess) {												\
    std::cerr << "CUDA driver error " << hipGetErrorString(__err) << " while calling " #__call << std::endl;		\
    std::cerr << "File: " << __FILE__ << ", function: " << __FUNCTION__ << ", line: " << __LINE__  << std::endl;	\
    throw std::runtime_error("CUDA driver error");									\
  }															\
} while(0)


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void sample_calibr_funct(const calibr_funct_t & calibr_funct);
std::vector<estim_event_t, aligned_allocator<estim_event_t>> contr_grid(const std::vector<PMT_data_t, aligned_allocator<PMT_data_t>> & PMT_data, calibr_funct_t calibr_funct);
__global__ void contr_grid_kernel(estim_event_t *estim_event_dev, PMT_data_t *PMT_data_dev, mdrf_spline_t *mdrf_dev, thresh_spline_t *thresh_dev, float *gain_dev);


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


int main(int argc, char **argv) {
  std::vector<estim_event_t, aligned_allocator<estim_event_t>> estim_event;
  std::vector<PMT_data_t, aligned_allocator<PMT_data_t>> PMT_data;
  calibr_funct_t calibr_funct;
  calibr_data_t calibr_data;
  
  calibr_data = get_calibration_data("../data/camera0_79x79_1.5mm_tc99m_mean", "../data/camera0_thresh.dat", "../data/camera0_79x79_1.5mm_tc99m_gains");
  calibr_funct = get_calibration_funct(calibr_data);
  sample_calibr_funct(calibr_funct);
  PMT_data = get_PMT_data("../data/ResPhantom022516-0mm_00.dat");
  estim_event = contr_grid(PMT_data, calibr_funct);
  write_estim_events(estim_event, "../data/estim_events_GPU.dat");
  return(0);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void sample_calibr_funct(const calibr_funct_t & calibr_funct) {
  const int num_sampl_x = 128;
  const int num_sampl_y = 128;
  std::array<std::array<float, num_sampl_y>, num_sampl_x> data;
  float pos_x[num_sampl_x];
  float pos_y[num_sampl_y];
  std::string filename;
  int nx, ny;
  int i, pmt;
  
  for(i = 0; i < num_sampl_x; ++i) {
    pos_x[i] = float(i) / float(num_sampl_x - 1);
  }
  for(i = 0; i < num_sampl_y; ++i) {
    pos_y[i] = float(i) / float(num_sampl_y - 1);
  }
  for(pmt = 0; pmt < NUM_PMTS; ++pmt) {
    for(nx = 0; nx < num_sampl_x; ++nx) {
      for(ny = 0; ny < num_sampl_y; ++ny) {
        data[nx][ny] = calibr_funct.mdrf[pmt](pos_x[nx], pos_y[ny]);
      }
    }
    std::ostringstream ss;
    ss << std::setw(3) << std::setfill('0') << pmt;
    filename = "../data/mdrf_samples_GPU_" + ss.str() + ".dat";
    write_dat_2d<float, num_sampl_x, num_sampl_y>(data, filename.c_str());
  }
  for(nx = 0; nx < num_sampl_x; ++nx) {
    for(ny = 0; ny < num_sampl_y; ++ny) {
      data[nx][ny] = calibr_funct.thresh(pos_x[nx], pos_y[ny]);
    }
  }
  write_dat_2d<float, num_sampl_x, num_sampl_y>(data, "../data/thresh_samples_GPU.dat");
  return;
}


std::vector<estim_event_t, aligned_allocator<estim_event_t>> contr_grid(const std::vector<PMT_data_t, aligned_allocator<PMT_data_t>> & PMT_data, calibr_funct_t calibr_funct) {
  std::vector<estim_event_t, aligned_allocator<estim_event_t>> estim_event(PMT_data.size());
  std::chrono::time_point<std::chrono::steady_clock> start, end;
  estim_event_t *estim_event_dev;
  thresh_spline_t *thresh_dev;
  PMT_data_t *PMT_data_dev;
  mdrf_spline_t *mdrf_dev;
  unsigned int num_events;
  float *gain_dev;
  
  num_events = (unsigned int) PMT_data.size();
  std::cout << "Number of events: " << num_events << "." << std::endl;
  start = std::chrono::steady_clock::now();
  if(num_events > 0) {
    CUDA_SAFE_CALL(hipMalloc(& PMT_data_dev, num_events * sizeof(PMT_data[0])));
    CUDA_SAFE_CALL(hipMalloc(& estim_event_dev, num_events * sizeof(estim_event[0])));
    CUDA_SAFE_CALL(hipMalloc(& mdrf_dev, NUM_PMTS * sizeof(*mdrf_dev)));
    CUDA_SAFE_CALL(hipMalloc(& thresh_dev, sizeof(*thresh_dev)));
    CUDA_SAFE_CALL(hipMalloc(& gain_dev, NUM_PMTS * sizeof(*gain_dev)));
    CUDA_SAFE_CALL(hipMemcpy(PMT_data_dev, PMT_data.data(), num_events * sizeof(PMT_data[0]), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(mdrf_dev, calibr_funct.mdrf, NUM_PMTS * sizeof(*mdrf_dev), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(thresh_dev, & calibr_funct.thresh, sizeof(*thresh_dev), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(gain_dev, calibr_funct.gain, NUM_PMTS * sizeof(*gain_dev), hipMemcpyHostToDevice));
    contr_grid_kernel<<<num_events, dim3(SIZE_CONTR_GRID, SIZE_CONTR_GRID)>>>(estim_event_dev, PMT_data_dev, mdrf_dev, thresh_dev, gain_dev);
    CUDA_SAFE_CALL(hipMemcpy(estim_event.data(), estim_event_dev, num_events * sizeof(estim_event[0]), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipFree(PMT_data_dev));
    CUDA_SAFE_CALL(hipFree(estim_event_dev));
    CUDA_SAFE_CALL(hipFree(mdrf_dev));
    CUDA_SAFE_CALL(hipFree(thresh_dev));
    CUDA_SAFE_CALL(hipFree(gain_dev));
  }
  end = std::chrono::steady_clock::now();
  std::chrono::duration<double> diff = end - start;
  std::cout << "Elapsed time: " << diff.count() << " s (" << double(num_events) / diff.count() << " events/s)." << std::endl;
  return(estim_event);
}


__global__ void contr_grid_kernel(estim_event_t *estim_event_dev, PMT_data_t *PMT_data_dev, mdrf_spline_t *mdrf_dev, thresh_spline_t *thresh_dev, float *gain_dev) {
  const unsigned int thread_index = threadIdx.y * SIZE_CONTR_GRID + threadIdx.x;
  __shared__ float log_like_values[SIZE_CONTR_GRID][SIZE_CONTR_GRID];
  __shared__ float tmp_data[NUM_PMTS];
  float log_like, max_log_like;
  int max_index_x, max_index_y;
  float current_x, current_y;
  unsigned int event_index;
  bool inside_x, inside_y;
  int index_x, index_y;
  float test_x, test_y;
  float camera_MDRF;
  int pmt, iter;
  float step;
  
  event_index = blockIdx.x;
  if(thread_index < NUM_PMTS) {
    tmp_data[thread_index] = PMT_data_dev[event_index].val[thread_index] / gain_dev[thread_index];
  }
  __syncthreads();
  current_x = current_y = float(1) / float(2);
  step = (float(1) - float(0)) / float(SIZE_CONTR_GRID);
  for(iter = 0; iter < NUM_CONTR_GRID_ITER; ++iter) {
    test_x = current_x + (float(threadIdx.x) - (float(SIZE_CONTR_GRID - 1) / 2.00f)) * step;
    test_y = current_y + (float(threadIdx.y) - (float(SIZE_CONTR_GRID - 1) / 2.00f)) * step;
    inside_x = (float(0) < test_x) && (test_x < float(1));
    inside_y = (float(0) < test_y) && (test_y < float(1));
    if(inside_x && inside_y) {
      log_like = float(0);
      for(pmt = 0; pmt < NUM_PMTS; ++pmt) {
        camera_MDRF = mdrf_dev[pmt](test_x, test_y);
        if((tmp_data[pmt] != float(0)) || (camera_MDRF != float(0))) {
          log_like += tmp_data[pmt] * logf(camera_MDRF) - camera_MDRF;
        }
      }
      log_like_values[threadIdx.x][threadIdx.y] = log_like;
    } else {
      log_like_values[threadIdx.x][threadIdx.y] = -HUGE_VALF;
    }
    __syncthreads();
    max_log_like = log_like_values[0][0];
    max_index_x = max_index_y = 0;
    for(index_x = 0; index_x < SIZE_CONTR_GRID; ++index_x) {
      for(index_y = 0; index_y < SIZE_CONTR_GRID; ++index_y) {
        if(max_log_like < log_like_values[index_x][index_y]) {
          max_log_like = log_like_values[index_x][index_y];
          max_index_x = index_x;
          max_index_y = index_y;
        }
      }
    }
    current_x = current_x + (float(max_index_x) - (float(SIZE_CONTR_GRID - 1) / 2.00f)) * step;
    current_y = current_y + (float(max_index_y) - (float(SIZE_CONTR_GRID - 1) / 2.00f)) * step;
    step /= CONTR_FACTOR;
  }
  if(thread_index == 0) {
    inside_x = (float(0) < current_x) && (current_x < float(1));
    inside_y = (float(0) < current_y) && (current_y < float(1));
    if(inside_x && inside_y) {
      log_like = max_log_like;
      for(pmt = 0; pmt < NUM_PMTS; ++pmt) {
        if(tmp_data[pmt] > float(0)) {
          log_like -= lgammaf(tmp_data[pmt] + float(1));
        }
      }
      estim_event_dev[event_index].valid = log_like > (*thresh_dev)(current_x, current_y);
      estim_event_dev[event_index].log_like = log_like;
    } else {
      estim_event_dev[event_index].valid = 0;
    }
    estim_event_dev[event_index].x_pos = CAMERA_MIN_POS + current_x * (CAMERA_MAX_POS - CAMERA_MIN_POS);
    estim_event_dev[event_index].y_pos = CAMERA_MIN_POS + current_y * (CAMERA_MAX_POS - CAMERA_MIN_POS);
  }
  return;
}
